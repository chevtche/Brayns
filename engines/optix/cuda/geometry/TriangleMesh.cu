#include "hip/hip_runtime.h"
/* Copyright (c) 2019, EPFL/Blue Brain Project
 * All rights reserved. Do not distribute without permission.
 *
 * This file is part of Brayns <https://github.com/BlueBrain/Brayns>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include "IntersectionRefinement.h"
#include <optix.h>
#include <optixu/optixu_aabb_namespace.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include "../../CommonStructs.h"

using namespace optix;

// This is to be plugged into an RTgeometry object to represent
// a triangle mesh with a vertex buffer of triangle soup (triangle list)
// with an interleaved position, normal, texturecoordinate layout.

rtBuffer<float3> vertices_buffer;
rtBuffer<float3> normal_buffer;
rtBuffer<float2> texcoord_buffer;
rtBuffer<int3> indices_buffer;

rtDeclareVariable(float2, texcoord, attribute texcoord, );
rtDeclareVariable(float3, v0, attribute v0, );
rtDeclareVariable(float3, v1, attribute v1, );
rtDeclareVariable(float3, v2, attribute v2, );
rtDeclareVariable(float2, t0, attribute t0, );
rtDeclareVariable(float2, t1, attribute t1, );
rtDeclareVariable(float2, t2, attribute t2, );
rtDeclareVariable(float2, ddx, attribute ddx, );
rtDeclareVariable(float2, ddy, attribute ddy, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

rtDeclareVariable(float3, back_hit_point, attribute back_hit_point, );
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(unsigned long, simulation_idx, attribute simulation_idx, );

static __device__ bool intersect_triangle_filtered(const Ray&    ray,
                                                   const float3& p0,
                                                   const float3& p1,
                                                   const float3& p2,
                                                   const float3& rayDx,
                                                   const float3& rayDy, 
                                                   float3& n,
                                                   float&  t,
                                                   float3&  beta,
                                                   float3&  gamma)
{
  const float3 e0 = p1 - p0;
  const float3 e1 = p0 - p2;
  n  = cross( e1, e0 );

  const float3 g0 = p0 - ray.origin;

  float3 e2 = ( 1.0f / dot( n, ray.direction ) ) * g0;
  float3 i  = cross( ray.direction, e2 );

  beta.x  = dot( i, e1 );
  gamma.x = dot( i, e0 );
  t     = dot( n, e2 );

  if(!((t<ray.tmax) & (t>ray.tmin) & (beta.x>=0.0f) & (gamma.x>=0.0f) & (beta.x+gamma.x<=1)))
      return false;

  float3 dirX = ray.direction + rayDx;
  e2 = ( 1.0f / dot( n, dirX ) ) * g0;
  i  = cross( dirX, e2 );
  beta.y  = dot( i, e1 );
  gamma.y = dot( i, e0 );

  float3 dirY = ray.direction + rayDy;
  e2 = ( 1.0f / dot( n, dirY ) ) * g0;
  i  = cross( dirY, e2 );
  beta.z  = dot( i, e1 );
  gamma.z = dot( i, e0 );

  return true; 
}

template <bool DO_REFINE>
static __device__ void meshIntersect(int primIdx)
{
    const int3 v_idx = indices_buffer[primIdx];

    const float3 p0 = vertices_buffer[v_idx.x];
    const float3 p1 = vertices_buffer[v_idx.y];
    const float3 p2 = vertices_buffer[v_idx.z];

    // Intersect ray with triangle
    float3 n;
    float t;
    float3 beta, gamma;
    if (intersect_triangle_filtered(ray, p0, p1, p2, prd.rayDx, prd.rayDy, n, t, beta, gamma))
    {
        if (rtPotentialIntersection(t))
        {
            v0 = p0;
            v1 = p1;
            v2 = p2;
            geometric_normal = normalize(n);
            if (normal_buffer.size() == 0)
                shading_normal = geometric_normal;
            else
            {
                float3 n0 = normal_buffer[v_idx.x];
                float3 n1 = normal_buffer[v_idx.y];
                float3 n2 = normal_buffer[v_idx.z];
                shading_normal = normalize(n1 * beta.x + n2 * gamma.x +
                                           n0 * (1.f - beta.x - gamma.x));
            }

            if (texcoord_buffer.size() == 0)
            {
                texcoord = make_float2(0.f, 0.f);
            }
            else
            {
                t0 = texcoord_buffer[v_idx.x];
                t1 = texcoord_buffer[v_idx.y];
                t2 = texcoord_buffer[v_idx.z];
                texcoord = t1 * beta.x + t2 * gamma.x +
                                       t0 * (1.f - beta.x - gamma.x);

                const float2 texcoordX = t1 * beta.y + t2 * gamma.y +
                                         t0 * (1.f - beta.y - gamma.y);
                const float2 texcoordY = t1 * beta.z + t2 * gamma.z +
                                         t0 * (1.f - beta.z - gamma.z);

                ddx = texcoordX - texcoord;
                ddy = texcoordY - texcoord;
            }

            if (DO_REFINE)
                refine_and_offset_hitpoint(ray.origin + t * ray.direction,
                                           ray.direction, geometric_normal, p0,
                                           back_hit_point, front_hit_point);
            simulation_idx = 0;
            rtReportIntersection(0);
        }
    }
}

RT_PROGRAM void intersect(int primIdx)
{
    meshIntersect<false>(primIdx);
}

RT_PROGRAM void intersect_refine(int primIdx)
{
    meshIntersect<true>(primIdx);
}

RT_PROGRAM void bounds(int primIdx, float result[6])
{
    const int3 v_idx = indices_buffer[primIdx];

    const float3 v0 = vertices_buffer[v_idx.x];
    const float3 v1 = vertices_buffer[v_idx.y];
    const float3 v2 = vertices_buffer[v_idx.z];
    const float area = length(cross(v1 - v0, v2 - v0));

    optix::Aabb* aabb = (optix::Aabb*)result;

    if (area > 0.0f && !isinf(area))
    {
        aabb->m_min = fminf(fminf(v0, v1), v2);
        aabb->m_max = fmaxf(fmaxf(v0, v1), v2);
    }
    else
    {
        aabb->invalidate();
    }
}
